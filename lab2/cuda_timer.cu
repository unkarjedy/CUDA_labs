#include "cuda_timer.hpp"

CudaTimer::CudaTimer(){
    hipEventCreate(&m_start);
    hipEventCreate(&m_stop);
}


void CudaTimer::start(){
    hipDeviceSynchronize();
    hipEventRecord(m_start, 0);
}

void CudaTimer::stop(){
    hipEventRecord(m_stop, 0);
    hipEventSynchronize(m_stop);
    //hipDeviceSynchronize();
}


float CudaTimer::getTime(){
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, m_start, m_stop);
    return elapsedTime;
}